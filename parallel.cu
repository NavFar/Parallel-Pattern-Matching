#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <assert.h>
#include <math.h>
#include "EasyBMP/EasyBMP.h"
#include <vector>
#include <cmath>
using namespace std;
// CUDA runtime
#include <hip/hip_runtime.h>
#include <>
#define BLOCK_SIZE 512
#define THRESHOLD 0.999
////////////////////////////////////////////////////////////
/*_  __                    _
 | |/ /___ _ __ _ __   ___| |___
 | ' // _ \ '__| '_ \ / _ \ / __|
 | . \  __/ |  | | | |  __/ \__ \
 |_|\_\___|_|  |_| |_|\___|_|___/
 */
////////////////////////////////////////////////////////////
__global__ void corr(RGBpixel *image,RGBpixel*pattern,uint* size,long patternMultRes)
{
        int overalIndex=blockIdx.x * blockDim.x +threadIdx.x;
        int imageIIndex=overalIndex/size[0];
        int imageJIndex=overalIndex%size[0];
        if(imageIIndex>=size[0]-size[2]||imageJIndex>=size[1]-size[3])
                return;
        long top=0;
        long bot=0;
        int sum=0;
        RGBpixel tempPat,tempImg;
        for(int i=0; i<size[2]; i++)
        {
                for(int j=0; j<size[3]; j++)
                {
                        sum++;
                        tempPat=pattern[i*size[3]+j];
                        tempImg=image[(imageIIndex+i)*size[1]+(imageJIndex+j)];
                        top+=((tempPat.Red   * tempImg.Red)+
                              (tempPat.Green * tempImg.Green)+
                              (tempPat.Blue  * tempImg.Blue))/3;
                        bot+=((tempImg.Red   * tempImg.Red)+
                              (tempImg.Green * tempImg.Green)+
                              (tempImg.Blue  * tempImg.Blue))/3;
                }
        }
        if((top/sqrt((float)(patternMultRes*bot)))>=THRESHOLD)
                printf("%d,%d\n",imageIIndex,imageJIndex);

}
////////////////////////////////////////////////////////////
__global__ void patternMultSum(RGBpixel * pattern, uint * size,long * result)
{
        extern __shared__ long load[];
        int overalIndex = blockIdx.x*blockDim.x+threadIdx.x;
        int j=overalIndex/size[0];
        int i=overalIndex%size[0];
        if(i>=size[0]||j>=size[1])
                load[threadIdx.x]=0;
        else{
                load[threadIdx.x]= ((pattern[overalIndex].Red*pattern[overalIndex].Red)+
                                    (pattern[overalIndex].Green*pattern[overalIndex].Green)+
                                    (pattern[overalIndex].Blue*pattern[overalIndex].Blue))/3;
        }
        __syncthreads();
        for(uint step=blockDim.x/2; step>0; step>>=1) {
                if(threadIdx.x<step)
                        load[threadIdx.x]+=load[threadIdx.x+step];
                __syncthreads();
        }
        if(threadIdx.x==0)
                result[blockIdx.x]=load[0];
}
////////////////////////////////////////////////////////////
void cudaDeviceWarmUp(int devID=0){
        hipSetDevice(devID);
        hipError_t error;
        hipDeviceProp_t deviceProp;
        error = hipGetDevice(&devID);
        if (error != hipSuccess)
        {
                printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_SUCCESS);
        }
        error = hipGetDeviceProperties(&deviceProp, devID);
        if (deviceProp.computeMode == hipComputeModeProhibited)
        {
                fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
                exit(EXIT_SUCCESS);
        }
        if (error != hipSuccess)
        {
                printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_SUCCESS);
        }
        else
        {
                printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
        }
}
////////////////////////////////////////////////////////////
void runCor(BMP pattern,BMP image, long multRes)
{
        hipError_t error;
        RGBpixel *patternData;
        int patternSize=pattern.TellWidth()*pattern.TellHeight();
        patternData = new RGBpixel[patternSize];
        for(int i=0; i<pattern.TellWidth(); i++)
        {
                for(int j=0; j<pattern.TellHeight(); j++)
                {
                        patternData[i*pattern.TellHeight()+j].Red=pattern(i,j)->Red;
                        patternData[i*pattern.TellHeight()+j].Green=pattern(i,j)->Green;
                        patternData[i*pattern.TellHeight()+j].Blue=pattern(i,j)->Blue;
                }
        }
        RGBpixel *d_patternData;
        error = hipMalloc((void **)&d_patternData, patternSize* sizeof (RGBpixel));
        if (error != hipSuccess)
        {
                printf("hipMalloc d_input returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        error = hipMemcpy(d_patternData, patternData, patternSize* sizeof (RGBpixel), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
                printf("hipMemcpy (d_patternData, patternData) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        RGBpixel *imageData;
        int imageSize=image.TellWidth()*image.TellHeight();
        int numberOfBlocks=(int)ceil((float)imageSize/BLOCK_SIZE);
        imageData = new RGBpixel[imageSize];
        for(int i=0; i<image.TellWidth(); i++)
        {
                for(int j=0; j<image.TellHeight(); j++)
                {
                        imageData[i*image.TellHeight()+j].Red=image(i,j)->Red;
                        imageData[i*image.TellHeight()+j].Green=image(i,j)->Green;
                        imageData[i*image.TellHeight()+j].Blue=image(i,j)->Blue;
                }
        }
        RGBpixel * d_imageData;
        error = hipMalloc((void **)&d_imageData, imageSize* sizeof (RGBpixel));
        if (error != hipSuccess)
        {
                printf("hipMalloc d_input returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        error = hipMemcpy(d_imageData, imageData, imageSize* sizeof (RGBpixel), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
                printf("hipMemcpy (d_patternData, patternData) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }

        uint size[]={(uint)image.TellWidth(),(uint)image.TellHeight(),(uint)pattern.TellWidth(),(uint)pattern.TellHeight()};
        uint *d_size;
        error = hipMalloc((void **)&d_size,  4*sizeof (uint));
        if (error != hipSuccess)
        {
                printf("hipMalloc d_input returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        error = hipMemcpy(d_size, size, 4*sizeof (uint), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
                printf("hipMemcpy (d_patternData, patternData) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        /////////////////////// grid and threads
        dim3 grid(numberOfBlocks,1,1);
        dim3 threads(BLOCK_SIZE,1,1);
        ///////////////////////
        hipEvent_t start;
        error = hipEventCreate(&start);
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }
        hipEvent_t stop;
        error = hipEventCreate(&stop);
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }
        error = hipEventRecord(start, NULL);
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }
        corr<<< grid, threads>>> (d_imageData,d_patternData,d_size,multRes);
        error = hipGetLastError();
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to launch kernel!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }

        // Record the stop event
        error = hipEventRecord(stop, NULL);

        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }

        // Wait for the stop event to complete
        error = hipEventSynchronize(stop);
        float msecTotal = 0.0f;
        error = hipEventElapsedTime(&msecTotal, start, stop);
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }
        hipFree(d_size);
        hipFree(d_imageData);
        hipFree(d_patternData);
        free(imageData);

}
////////////////////////////////////////////////////////////
long runPatternMult(BMP pattern)
{
        hipError_t error;
        RGBpixel *patternData;
        int patternSize=pattern.TellWidth()*pattern.TellHeight();
        int numberOfBlocks=(int)ceil((float)patternSize/BLOCK_SIZE);
        patternData = new RGBpixel[patternSize];
        for(int i=0; i<patternSize; i++) {
                patternData[i].Red=pattern(i%pattern.TellWidth(),i/pattern.TellWidth())->Red;
                patternData[i].Green=pattern(i%pattern.TellWidth(),i/pattern.TellWidth())->Green;
                patternData[i].Blue=pattern(i%pattern.TellWidth(),i/pattern.TellWidth())->Blue;
        }
        RGBpixel *d_patternData;
        error = hipMalloc((void **)&d_patternData, patternSize* sizeof (RGBpixel));
        if (error != hipSuccess)
        {
                printf("hipMalloc d_input returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        error = hipMemcpy(d_patternData, patternData, patternSize* sizeof (RGBpixel), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
                printf("hipMemcpy (d_patternData, patternData) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        long *d_patternSum;
        error = hipMalloc((void **)&d_patternSum, numberOfBlocks* sizeof (long));
        if (error != hipSuccess)
        {
                printf("hipMalloc d_input returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        uint patternDim[2];
        patternDim[0]=pattern.TellWidth();
        patternDim[1]=pattern.TellHeight();
        uint * d_patternDim;
        error = hipMalloc((void **)&d_patternDim, 2* sizeof (uint));
        if (error != hipSuccess)
        {
                printf("hipMalloc d_input returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        error = hipMemcpy(d_patternDim, patternDim, 2* sizeof (uint), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
                printf("hipMemcpy (d_patternData, patternData) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        /////////////////////// grid and threads
        dim3 grid(numberOfBlocks,1,1);
        dim3 threads(BLOCK_SIZE,1,1);
        ///////////////////////
        hipEvent_t start;
        error = hipEventCreate(&start);
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }
        hipEvent_t stop;
        error = hipEventCreate(&stop);
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }
        error = hipEventRecord(start, NULL);
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }
        patternMultSum<<< grid, threads,BLOCK_SIZE * sizeof(long)>>> (d_patternData,d_patternDim,d_patternSum);
        error = hipGetLastError();
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to launch kernel!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }

        // Record the stop event
        error = hipEventRecord(stop, NULL);

        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }

        // Wait for the stop event to complete
        error = hipEventSynchronize(stop);
        float msecTotal = 0.0f;
        error = hipEventElapsedTime(&msecTotal, start, stop);
        if (error != hipSuccess)
        {
                fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
                exit(EXIT_FAILURE);
        }
        long * patternSum;
        patternSum= new long[numberOfBlocks];
        error = hipMemcpy(patternSum, d_patternSum, numberOfBlocks * sizeof (long), hipMemcpyDeviceToHost);
        if (error != hipSuccess)
        {
                printf("hipMemcpy (h_output,d_input) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
        }
        long sum=0;
        for(int i=0; i<numberOfBlocks; i++)
                sum+=patternSum[i];
        // cout<<sum<<endl;
        hipFree(d_patternData);
        hipFree(d_patternDim);
        hipFree(d_patternSum);
        free(patternSum);
        free(patternData);
        return sum;

}
// void runCor(RGBpixel * d_pattern,BMP image, long multRes,uint patternWidth, uint patternHeight)

////////////////////////////////////////////////////////////
int main()
{
        cudaDeviceWarmUp();
        BMP pattern,rPattern,image;
        long patternMultSum=0;
        image.ReadFromFile("Inputs/collection.bmp");
        pattern.ReadFromFile("Inputs/collection_coin.bmp");
        patternMultSum=runPatternMult(pattern);
        runCor(pattern,image,patternMultSum);
        rotateImage(pattern, rPattern);
        runCor(rPattern,image,patternMultSum);
}
